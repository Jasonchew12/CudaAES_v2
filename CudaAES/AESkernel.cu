﻿// main.cpp
#include "AES.cuh"

#include <chrono>
#include <cstdlib>

void DisplayAESExplanationkey(unsigned char* key, enum keySize size)
{
    const int expandedKeySizeDisplay = 240;

    // the expanded key
    unsigned char expandedKeyDisplay[expandedKeySizeDisplay];


    CreateExpandKey(expandedKeyDisplay, key, size, expandedKeySizeDisplay);

    std::cout << "Expanded Key:\n";
    for (int i = 0; i < expandedKeySizeDisplay; i++) {
        // Print the block number at the beginning of each new line
        if (i % 16 == 0) {
            std::cout << (i / 16 + 1) << ": ";  // Block number starts from 1
        }

        std::cout << std::hex << std::setw(2) << std::setfill('0')
            << static_cast<int>(expandedKeyDisplay[i]);

        // Insert a space after each byte for readability
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;  // Print a newline after every 16 bytes
        }
        else {
            std::cout << " ";  // Print a space otherwise
        }
    }

}

#pragma region CUDAFILEVersion
int main(int argc, char* argv[])
{
    // Define file paths for input and output
    //std::string inputFilePath = "FileToEncrypt/500mb-examplefile-com.txt";
    //           // Input file to encrypt
    std::string inputFilePath = "FileToEncrypt/500mb.txt";           // Input file to encrypt
    std::string encryptedFilePath = "EncryptFile/500mbEncryptedCUDA.bin"; // Encrypted output file
    std::string decryptedFilePath = "DecryptFile/500mb_unencrypted.txt"; // Decrypted output file

    // Define the key and key size (256-bit in this example)
    unsigned char key[32] = { "HalloWorldThisIsAStringOfLetter" }; // Example key
    enum keySize size = SIZE_32;  // You can change this to SIZE_16 or SIZE_24 for 128-bit or 192-bit keys

    // Display the expanded key for informational purposes
    DisplayAESExplanationkey(key, size);

    // Measure encryption time
    auto encryptionStart = std::chrono::high_resolution_clock::now();

    // Encrypt the file
    if (EncryptFile(inputFilePath, encryptedFilePath, key, size)) {
        std::cout << "File encryption completed successfully!" << std::endl;
    }
    else {
        std::cerr << "File encryption failed!" << std::endl;
        return -1;
    }

    auto encryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> encryptionDuration = encryptionEnd - encryptionStart;
    std::cout << "Time taken for encryption: " << encryptionDuration.count() << " seconds" << std::endl;

    // Measure decryption time
    auto decryptionStart = std::chrono::high_resolution_clock::now();

    // Decrypt the file
    if (DecryptFile(encryptedFilePath, decryptedFilePath, key, size)) {
        std::cout << "File decryption completed successfully!" << std::endl;
    }
    else {
        std::cerr << "File decryption failed!" << std::endl;
        return -1;
    }

    auto decryptionEnd = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> decryptionDuration = decryptionEnd - decryptionStart;
    std::cout << "Time taken for decryption: " << decryptionDuration.count() << " seconds" << std::endl;

    // Optionally, you can open the original and decrypted files for comparison
    std::string openOriginalFileCommand = "start " + inputFilePath;
    system(openOriginalFileCommand.c_str());

    std::string openDecryptedFileCommand = "start " + decryptedFilePath;
    system(openDecryptedFileCommand.c_str());

    return 0;
}
#pragma endregion

#pragma region CUDASmallAES
//int main() {
//    // Define a small input message (exactly 16 bytes, no padding required)
//    const char* testInput = "abcdef1234567890";  // 16 bytes (exactly one AES block)
//    unsigned char encrypted[16];
//    unsigned char decrypted[16];
//    
//    // AES key (this should match the key size you're using in the real test)
//    unsigned char key[32] = { "Hallo World" };  // Example key, AES-256 (32 bytes)
//
//    // Print the original message
//    std::cout << "Original message: " << testInput << std::endl;
//
//    // Encrypt the message
//    char resultEncrypt = AES_Encrypt((unsigned char*)testInput, encrypted, key, SIZE_32);
//    if (resultEncrypt != SUCCESS) {
//        std::cerr << "Encryption failed with error code: " << resultEncrypt << std::endl;
//        return -1;
//    }
//
//    // Print encrypted message in hex
//    std::cout << "Encrypted message (hex): ";
//    for (int i = 0; i < 16; ++i) {
//        std::cout << std::hex << (int)encrypted[i] << " ";
//    }
//    std::cout << std::endl;
//
//    // Decrypt the message
//    char resultDecrypt = AES_Decrypt(encrypted, decrypted, key, SIZE_32);
//    if (resultDecrypt != SUCCESS) {
//        std::cerr << "Decryption failed with error code: " << resultDecrypt << std::endl;
//        return -1;
//    }
//
//    // Print the decrypted message
//    std::cout << "Decrypted message: " << decrypted << std::endl;
//
//    // Compare original and decrypted message
//    if (std::memcmp(testInput, decrypted, 16) == 0) {
//        std::cout << "Decryption successful!" << std::endl;
//    } else {
//        std::cout << "Decryption failed. Original and decrypted message do not match." << std::endl;
//    }
//
//    return 0;
//}
#pragma endregion



